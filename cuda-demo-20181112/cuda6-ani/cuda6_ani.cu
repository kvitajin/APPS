#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"
#include "animation.h"

// Demo kernel to create chess board
__global__ void kernel_creategradient( CUDA_Pic colorPic )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	int dy = x * colorPic.Size.y / colorPic.Size.x + y - colorPic.Size.y;
	unsigned char color = 255 * abs( dy ) / colorPic.Size.y;

	uchar3 bgr = ( dy < 0 ) ? ( uchar3 ) { color, 255 - color, 0 } : ( uchar3 ) { 0, 255 - color, color };

	// Store point into image
	colorPic.P_uchar3[ y * colorPic.Size.x + x ] = bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= smallPic.Size.y ) return;
	if ( x >= smallPic.Size.x ) return;
	int by = y + position.y;
	int bx = x + position.x;
	if ( by >= bigPic.Size.y || by < 0 ) return;
	if ( bx >= bigPic.Size.x || bx < 0 ) return;

	// Get point from small image
	uchar4 fg_bgra = smallPic.P_uchar4[ y * smallPic.Size.x + x ];
	uchar3 bg_bgr = bigPic.P_uchar3[ by * bigPic.Size.x + bx ];
	uchar3 bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	bgr.x = fg_bgra.x * fg_bgra.w / 255 + bg_bgr.x * ( 255 - fg_bgra.w ) / 255;
	bgr.y = fg_bgra.y * fg_bgra.w / 255 + bg_bgr.y * ( 255 - fg_bgra.w ) / 255;
	bgr.z = fg_bgra.z * fg_bgra.w / 255 + bg_bgr.z * ( 255 - fg_bgra.w ) / 255;

	// Store point into image
	bigPic.P_uchar3[ by * bigPic.Size.x + bx ] = bgr;
}

void cu_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 32;
	dim3 blocks( ( smallPic.Size.x + block_size - 1 ) / block_size, ( smallPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_insertimage<<< blocks, threads >>>( bigPic, smallPic, position );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

void Animation::start( CUDA_Pic bgPic, CUDA_Pic insPic )
{
	if ( initialized ) return;
	hipError_t cerr;

	cuda_bgPic = bgPic;
	cuda_resPic = bgPic;
	cuda_insPic = insPic;

	// Memory allocation in GPU device
	// Memory for background
	cerr = hipMalloc( &cuda_bgPic.P_void, cuda_bgPic.Size.x * cuda_bgPic.Size.y * sizeof( uchar3 ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Memory for inserted image
	cerr = hipMalloc( &cuda_insPic.P_void, cuda_insPic.Size.x * cuda_insPic.Size.y * sizeof( uchar4 ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Memory for result
	cerr = hipMalloc( &cuda_resPic.P_void, cuda_resPic.Size.x * cuda_resPic.Size.y * sizeof( uchar3 ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Creation of background gradient
	int block_size = 32;
	dim3 blocks( ( cuda_bgPic.Size.x + block_size - 1 ) / block_size, ( cuda_bgPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_creategradient<<< blocks, threads >>>( cuda_bgPic );

	// Copy data to GPU device
	cerr = hipMemcpy( cuda_insPic.P_void, insPic.P_void, cuda_insPic.Size.x * cuda_insPic.Size.y * sizeof( uchar4 ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	initialized = 1;
}

void Animation::next( CUDA_Pic resPic, int2 position )
{
	if ( !initialized ) return;

	hipError_t cerr;

	// Copy data internally GPU from background into result
	cerr = hipMemcpy( cuda_resPic.P_void, cuda_bgPic.P_void, cuda_bgPic.Size.x * cuda_bgPic.Size.y * sizeof( uchar3 ), hipMemcpyDeviceToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// insert picture
	int block_size = 32;
	dim3 blocks( ( cuda_insPic.Size.x + block_size - 1 ) / block_size, ( cuda_insPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_insertimage<<< blocks, threads >>>( cuda_resPic, cuda_insPic, position );

	// Copy data to GPU device
	cerr = hipMemcpy( resPic.P_void, cuda_resPic.P_void, cuda_resPic.Size.x * cuda_resPic.Size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

}

void Animation::stop()
{
	if ( !initialized ) return;

	hipFree( cuda_bgPic.P_void );
	hipFree( cuda_resPic.P_void );
	hipFree( cuda_insPic.P_void );

	initialized = 0;
}


