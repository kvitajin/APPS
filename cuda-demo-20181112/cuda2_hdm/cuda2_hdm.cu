// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology usage without unified memory.
//
// Multiplication of elements in float array.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *pole, int L, float Mult )
{
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	// if grid is greater then length of array...
	if ( l >= L ) return;

	pole[ l ] *= Mult;
}

void cu_run_mult( float *Array, int Length, float Mult )
{
	hipError_t cerr;
	int threads = 128;
	int blocks = ( Length + threads - 1 ) / threads;

	// Memory allocation in GPU device
	float *hipArray;
	cerr = hipMalloc( &hipArray, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Copy data from PC to GPU device
	cerr = hipMemcpy( hipArray, Array, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation
	kernel_mult<<< blocks, threads >>>( hipArray, Length, Mult );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( Array, hipArray, Length * sizeof( float ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Free memory
	hipFree( hipArray );
}
