#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "pic_type.h"

// Every threads identifies its position in grid and in block and modify image
__global__ void kernel_animation( CUDA_Pic cuda_pic )
{
	// X,Y coordinates 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( x >= cuda_pic.Size.x ) return;
	if ( y >= cuda_pic.Size.y ) return;

	// Point [x,y] selection from image
	uchar3 bgr, tmp = cuda_pic.P_uchar3[ y * cuda_pic.Size.x + x ];

	// color rotation
    bgr.x = tmp.y;
    bgr.y = tmp.z;
    bgr.z = tmp.x;

	// Store point [x,y] back to image
	cuda_pic.P_uchar3[ y * cuda_pic.Size.x + x ] = bgr;
}

void cu_run_animation( CUDA_Pic pic, uint2 block_size )
{
	hipError_t cerr;

	// Grid creation with computed organization
	dim3 grid( ( pic.Size.x + block_size.x - 1 ) / block_size.x, ( pic.Size.y + block_size.y - 1 ) / block_size.y );
	kernel_animation<<< grid, dim3( block_size.x, block_size.y ) >>>( pic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();

}
