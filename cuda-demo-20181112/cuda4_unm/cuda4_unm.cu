#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CUDA_Pic colorPic, CUDA_Pic bwPic )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	// Get point from color picture
	uchar3 bgr = colorPic.P_uchar3[ y * colorPic.Size.x + x ];

	// Store BW point to new image
	bwPic.P_uchar1[ y * bwPic.Size.x + x ].x = bgr.x * 0.11 + bgr.y * 0.59 + bgr.z * 0.30;
}

void cu_run_grayscale( CUDA_Pic colorPic, CUDA_Pic bwPic )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 16;
	dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_grayscale<<< blocks, threads >>>( colorPic, bwPic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}
