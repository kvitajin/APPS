// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage.
// Global variables usage in threads, use of printf.
//
// Every thread displays information of its position in block,
// position of block in grid and global position.
//
// ***********************************************************************


#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

// Demo kernel will display all global variables of grid organization.
// Warning! Function printf is available from compute capability 2.x
__global__ void thread_hierarchy()
{
    // Global variables
    // Grid dimension -				gridDim
	// Block position in grid -		blockIdx
	// Block dimension -			blockDim
	// Thread position in block -	threadIdx
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    printf( "Block{%d,%d}[%d,%d] Thread{%d,%d}[%d,%d] [%d,%d]\n",
	    gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
		blockDim.x, blockDim.y, threadIdx.x, threadIdx.y, x, y );
}

void cu_run_cuda( dim3 grid_size, dim3 block_size )
{
	hipError_t cerr;
	// Following command can increase internal buffer for printf function
    /*cerr = cudaDeviceSetLimit( cudaLimitPrintfFifoSize, required_size );
	if ( err != cudaSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, cudaGetErrorString( cerr ) );
    */

	// Thread creation from selected kernel:
	// first parameter dim3 is grid dimension
	// second parameter dim3 is block dimension
    thread_hierarchy<<< grid_size, block_size >>>();

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Output from printf is in GPU memory. 
	// To get its contens it is necessary to synchronize device.

	hipDeviceSynchronize();
}
