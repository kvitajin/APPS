#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Manipulation with prepared image.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"

// Every threads identifies its position in grid and in block and modify image
__global__ void kernel_animation( CUDA_Pic cuda_pic )
{
	// X,Y coordinates 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( x >= cuda_pic.Size.x ) return;
	if ( y >= cuda_pic.Size.y ) return;

	// Point [x,y] selection from image
	uchar3 bgr, tmp = cuda_pic.P_uchar3[ y * cuda_pic.Size.x + x ];

	// color rotation
    bgr.x = tmp.y;
    bgr.y = tmp.z;
    bgr.z = tmp.x;

	// Store point [x,y] back to image
	cuda_pic.P_uchar3[ y * cuda_pic.Size.x + x ] = bgr;
}

void cu_run_animation( CUDA_Pic pic, uint2 block_size )
{
	hipError_t cerr;

	CUDA_Pic cudaPic;
	cudaPic.Size = pic.Size;

	// Memory allocation in GPU device
	cerr = hipMalloc( &cudaPic.P_void, cudaPic.Size.x * cudaPic.Size.y * sizeof( uchar3 ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Copy data to GPU device
	cerr = hipMemcpy( cudaPic.P_void, pic.P_void, cudaPic.Size.x * cudaPic.Size.y * sizeof( uchar3 ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation with computed organization
	dim3 mrizka( ( cudaPic.Size.x + block_size.x - 1 ) / block_size.x, ( cudaPic.Size.y + block_size.y - 1 ) / block_size.y );
	kernel_animation<<< mrizka, dim3( block_size.x, block_size.y ) >>>( cudaPic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( pic.P_void, cudaPic.P_void, pic.Size.x * pic.Size.y * sizeof( uchar3 ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Free memory
	hipFree( cudaPic.P_void );

	hipDeviceSynchronize();
}
