#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include "pic_type.h"

// Demo kernel to create chess board
__global__ void kernel_chessboard( CUDA_Pic colorPic )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	unsigned char borw = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

	// Store point into image
	colorPic.P_uchar3[ y * colorPic.Size.x + x ] = { borw, borw, borw };
}

void cu_create_chessboard( CUDA_Pic colorPic, int square_size )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 blocks( ( colorPic.Size.x + square_size - 1 ) / square_size, ( colorPic.Size.y + square_size - 1 ) / square_size );
	dim3 threads( square_size, square_size );
	kernel_chessboard<<< blocks, threads >>>( colorPic );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CUDA_Pic colorPic, uchar3 color )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= colorPic.Size.y ) return;
	if ( x >= colorPic.Size.x ) return;

	int diagonal = sqrtf( colorPic.Size.x * colorPic.Size.x + colorPic.Size.y * colorPic.Size.y );
	int dx = x - colorPic.Size.x / 2;
	int dy = y - colorPic.Size.y / 2;
	int dxy = sqrtf( dx * dx + dy * dy ) - diagonal / 2;

	// Store point into image
	colorPic.P_uchar4[ y * colorPic.Size.x + x ] =
		{ color.x, color.y, color.z, ( unsigned char ) ( 255 - 255 * dxy / ( diagonal / 2 ) ) };
}

void cu_create_alphaimg( CUDA_Pic colorPic, uchar3 color )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 32;
	dim3 blocks( ( colorPic.Size.x + block_size - 1 ) / block_size, ( colorPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_alphaimg<<< blocks, threads >>>( colorPic, color );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
	// X,Y coordinates and check image dimensions
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( y >= smallPic.Size.y ) return;
	if ( x >= smallPic.Size.x ) return;
	int by = y + position.y;
	int bx = x + position.x;
	if ( by >= bigPic.Size.y || by < 0 ) return;
	if ( bx >= bigPic.Size.x || bx < 0 ) return;

	// Get point from small image
	uchar4 fg_bgra = smallPic.P_uchar4[ y * smallPic.Size.x + x ];
	uchar3 bg_bgr = bigPic.P_uchar3[ by * bigPic.Size.x + bx ];
	uchar3 bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	bgr.x = fg_bgra.x * fg_bgra.w / 255 + bg_bgr.x * ( 255 - fg_bgra.w ) / 255;
	bgr.y = fg_bgra.y * fg_bgra.w / 255 + bg_bgr.y * ( 255 - fg_bgra.w ) / 255;
	bgr.z = fg_bgra.z * fg_bgra.w / 255 + bg_bgr.z * ( 255 - fg_bgra.w ) / 255;

	// Store point into image
	bigPic.P_uchar3[ by * bigPic.Size.x + bx ] = bgr;
}

void cu_insertimage( CUDA_Pic bigPic, CUDA_Pic smallPic, int2 position )
{
	hipError_t cerr;

	// Grid creation, size of grid must be equal or greater than images
	int block_size = 32;
	dim3 blocks( ( smallPic.Size.x + block_size - 1 ) / block_size, ( smallPic.Size.y + block_size - 1 ) / block_size );
	dim3 threads( block_size, block_size );
	kernel_insertimage<<< blocks, threads >>>( bigPic, smallPic, position );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	hipDeviceSynchronize();
}
